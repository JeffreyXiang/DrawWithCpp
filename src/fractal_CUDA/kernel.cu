#include "hip/hip_runtime.h"
#include "kernel.h"
#include "stdio.h"
#include "stdlib.h"

#include "complex.h"

__device__ int max_Interations = 1024;

__global__ void MandelbrotKernel(unsigned char* img, int width, int height, double xMin, double yMin, double xMax, double yMax)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= width * height) return;
    int xIdx = index % width;
    int yIdx = index / width;
    double x = xMin * (width - xIdx) / width + xMax * xIdx / width;
    double y = yMin * (height - yIdx) / height + yMax * yIdx / height;
    double2 z = { 0, 0 };
    double2 c = { x, y };
    for (int i = 0; i < max_Interations; i++)
    {
        z = complexPlus(complexSquare(z), c);
        if (complexLength2(z) > 4.0)
        {
            img[index] = i % 256;
            break;
        }
    }
    if (index % (int)(width * height / 100) == 0)
    {
        printf(".");
    }
}

__global__ void JuliaKernel(unsigned char* img, int width, int height, double cx, double cy, double xMin, double yMin, double xMax, double yMax)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int xIdx = index % width;
    int yIdx = index / width;
    double x = xMin * (width - xIdx) / width + xMax * xIdx / width;
    double y = yMin * (height - yIdx) / height + yMax * yIdx / height;
    double2 z = { x, y };
    double2 c = { cx, cy };
    for (int i = 0; i < max_Interations; i++)
    {
        z = complexPlus(complexSquare(z), c);
        if (complexLength2(z) > 4.0)
        {
            img[index] = i % 256;
            break;
        }
    }
    if (index % (int)(width * height / 100) == 0)
    {
        printf(".");
    }
}

void Mandelbrot(unsigned char* img, int width, int height, double xMin, double yMin, double xMax, double yMax)
{
    MandelbrotKernel << <ceil(width * height / 1024), 1024 >> > (img, width, height, xMin, yMin, xMax, yMax);

}

void Julia(unsigned char* img, int width, int height, double cx, double cy, double xMin, double yMin, double xMax, double yMax)
{
    JuliaKernel << <ceil(width * height / 1024), 1024 >> > (img, width, height, cx, cy, xMin, yMin, xMax, yMax);
}
