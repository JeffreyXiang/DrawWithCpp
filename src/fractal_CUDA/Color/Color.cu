#include "hip/hip_runtime.h"
#include ""
#include <stdint.h>

#include "Color.h"

__device__ CU_Color CU_Color_mix(CU_Color C1, CU_Color C2, double k)
{
    CU_Color res;
    k = k < 0 ? 0 : k > 1 ? 1 : k;
    res = { k * C1.red + (1 - k) * C2.red, k * C1.green + (1 - k) * C2.green, k * C1.blue + (1 - k) * C2.blue, k * C1.alpha + (1 - k) * C2.alpha };
    return res;
}

__device__ CU_Color CU_Color_overlay(CU_Color C1, CU_Color C2)
{
    CU_Color res;
    double alpha = 1 - (1 - C1.alpha) * (1 - C2.alpha);
    res = { (C1.alpha * C1.red + (1 - C1.alpha) * C2.alpha * C2.red) / alpha,
            (C1.alpha * C1.green + (1 - C1.alpha) * C2.alpha * C2.green) / alpha,
            (C1.alpha * C1.blue + (1 - C1.alpha) * C2.alpha * C2.blue) / alpha,
            alpha };
    return res;
}
